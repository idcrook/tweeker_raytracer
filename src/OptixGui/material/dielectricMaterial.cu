#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "material.cuh"

// Ray state variables
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload,  );

// "Global" variables
rtDeclareVariable(rtObject, sysWorld, , );

// The point and normal of intersection
rtDeclareVariable(HitRecord, hitRecord, attribute hitRecord, );

// Material variables
rtDeclareVariable(float, eta, , );

inline __device__ float fresnelSchlick(
    const float cosThetaI, const float etaI, const float etaT)
{
    float r0 = (etaI-etaT) / (etaI+etaT);
    r0 = r0*r0;
    return r0 + (1.f-r0)*powf((1.f-cosThetaI), 5.f);
}

inline __device__ float3 emitted() {
    return make_float3(0.f, 0.f, 0.f);
}

inline __device__ float scatteringPdf(){
  return false;
}

RT_PROGRAM void closestHit()
{
    // Get the ray's unit direction
    float3 unitDirection = optix::normalize(theRay.direction);

    // Determine if inside or outside of object
    float3 localNormal;
    float etaI, etaT;
    if (optix::dot(theRay.direction, hitRecord.normal) < 0.0f)
    {
        // Outside the object
        localNormal = hitRecord.normal;
        etaI = 1.0f;
        etaT = eta;
    }
    else
    {
        // Inside the object
        localNormal = -hitRecord.normal;
        etaI = eta;
        etaT = 1.0f;
    }

    // Snell's Law
    //  etaI * sinThetaI = etaT * sinThetaT
    // If
    //  (etaI/etaT) * sinThetaI > 1.0
    // Then
    //  sinThetaT does not exist, and no transmission is possible
    float3 scatterDirection;
    float cosThetaI =
        optix::min(optix::dot(-unitDirection, localNormal), 1.0f);
    float sinThetaI = sqrtf(1.0f - cosThetaI*cosThetaI);
    if ( ((etaI/etaT)*sinThetaI) > 1.0f )
    {
        // No Transmission is possible
        scatterDirection = optix::reflect(unitDirection, localNormal);
    }
    else
    {
        // Transmission + Reflection
        float reflectProb = fresnelSchlick(cosThetaI, etaI, etaT);
        if (randf(thePrd.seed) < reflectProb)
        { // Reflection
            scatterDirection = optix::reflect(unitDirection, localNormal);
        }
        else
        { // Transmission
            float sinThetaT = optix::min((etaI/etaT)*sinThetaI, 1.0f);
            float cosThetaT = sqrtf(1.0f - sinThetaT*sinThetaT);
            scatterDirection =
                (etaI/etaT)*(unitDirection + cosThetaI*localNormal) -
                cosThetaT*localNormal;
        }
    }

    // if(cosThetaI > 1.0f)
    //     printf("costThetaI is greater than unity: %f", cosThetaI);
    // if(!(sinThetaI == sinThetaI))
    //     printf("sinThetaI is NaN: %f", sinThetaI);

    thePrd.emitted = emitted();
    thePrd.is_specular = true;
    thePrd.materialType = Dielectric;
    thePrd.scatterEvent = Ray_Hit;
    thePrd.scattered_origin = hitRecord.point;
    thePrd.scattered_direction = scatterDirection;
    // thePrd.attenuation = make_float3(0.8f, 0.85f, 0.82f); // for greenish glass
    thePrd.attenuation = make_float3(1.0f, 1.0f, 1.0f);
    thePrd.scattered_pdf = scatteringPdf();
    thePrd.hit_normal = hitRecord.normal;
}
