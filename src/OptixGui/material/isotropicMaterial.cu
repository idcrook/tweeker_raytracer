#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "material.cuh"

// Ray state variables
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload,  );

// "Global" variables
rtDeclareVariable(rtObject, sysWorld, , );

// The point and normal of intersection
rtDeclareVariable(HitRecord, hitRecord, attribute hitRecord, );

// Material variables
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3)>, sampleTexture, , );

inline __device__ float3 emitted() {
    return make_float3(0.f, 0.f, 0.f);
}

inline __device__ float scatteringPdf() {
  return false;
}

RT_PROGRAM void closestHit()
{
    float3 scatterDirection = randomInUnitSphere(thePrd.seed);

    thePrd.emitted = emitted();
    thePrd.is_specular = true; // ???
    thePrd.materialType = Isotropic;

    thePrd.scatterEvent = Ray_Hit;
    thePrd.hit_normal = hitRecord.normal;
    thePrd.scattered_origin = hitRecord.point;
    thePrd.scattered_direction = scatterDirection;
    thePrd.attenuation = sampleTexture(hitRecord.u, hitRecord.v, hitRecord.point);
    thePrd.scattered_pdf = scatteringPdf();
}
