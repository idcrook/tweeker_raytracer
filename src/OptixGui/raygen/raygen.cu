#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../lib/raydata.cuh"
#include "../lib/random.cuh"
#include "../scene/camera.cuh"

#include "../pdf/pdf.cuh"

// using namespace optix;

// Optix program built-in indices
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );

// Ray state variables
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload,  );

// "Global" variables
rtDeclareVariable(rtObject, sysWorld, , );
rtBuffer<float3, 2> sysOutputBuffer;

// Ray Generation variables
rtDeclareVariable(int, numSamples, , );
rtDeclareVariable(int, maxRayDepth, , );

// "sky" illumination for misses
rtDeclareVariable(int, skyLight, , );


// PDF callable programs
rtDeclareVariable(rtCallableProgramId<float(pdf_in&)>, value, , );
rtDeclareVariable(rtCallableProgramId<float3(pdf_in&, uint32_t&)>, generate, , );

inline __device__ float3 removeNaNs(float3 radiance)
{
    float3 r = radiance;
    if(!(r.x == r.x)) r.x = 0.0f;
    if(!(r.y == r.y)) r.y = 0.0f;
    if(!(r.z == r.z)) r.z = 0.0f;
    return r;
}

inline __device__ float3 missColor(const optix::Ray &theRay)
{
    if (skyLight) {
        float3 unitDirection = optix::normalize(theRay.direction);
        float t = 0.5f * (unitDirection.y + 1.0f);
        // "sky" gradient
        float3 missColor = (1.0f-t) * make_float3(1.0f, 1.0f, 1.0f)
            + t * make_float3(0.5f, 0.7f, 1.0f);
        return missColor;
    } else {
        return make_float3(0.0f); // darkness in the void
    }
}


inline __device__ float3 color(optix::Ray& theRay, uint32_t& seed)
{
    PerRayData thePrd;
    thePrd.seed = seed;
    float3 sampleRadiance = make_float3(1.0f, 1.0f, 1.0f);
    thePrd.gatherTime = cameraTime0 + randf(seed)*(cameraTime1 - cameraTime0);

    for(int i = 0; i < maxRayDepth; i++)
    {
        rtTrace(sysWorld, theRay, thePrd);
        if (thePrd.scatterEvent == Ray_Miss)
        {
            return sampleRadiance * missColor(theRay);
        }
        else if (thePrd.scatterEvent == Ray_Finish)
        {
            return sampleRadiance * thePrd.attenuation;
        }
        else if (thePrd.scatterEvent == Ray_Cancel)
        {
            return sampleRadiance * thePrd.emitted;
        }
        else { // ray is still alive, and got properly bounced
            if (thePrd.is_specular) {
                sampleRadiance = sampleRadiance * thePrd.attenuation;
                theRay = optix::make_Ray(
                    thePrd.scattered_origin,
                    thePrd.scattered_direction,
                    0,
                    1e-3f,
                    RT_DEFAULT_MAX
                    );

            } else {
                pdf_in in(thePrd.scattered_origin, thePrd.scattered_direction, thePrd.hit_normal);
                float3 pdf_direction = generate(in, seed);
                float pdf_val = value(in);

                // sampleRadiance = optix::clamp(thePrd.emitted +
                //                               (thePrd.attenuation * thePrd.scattered_pdf * sampleRadiance) / pdf_val,
                //                               0.f, 1.f);

                sampleRadiance = thePrd.emitted +
                    (thePrd.attenuation * thePrd.scattered_pdf * sampleRadiance) / pdf_val;

                theRay = optix::make_Ray(/* origin   : */ in.origin,
                                         /* direction: */ pdf_direction,
                                         /* ray type : */ 0,
                                         /* tmin     : */ 1e-3f,
                                         /* tmax     : */ RT_DEFAULT_MAX);
            }
        }
    }
    seed = thePrd.seed;

    return make_float3(0.0f);
}

RT_PROGRAM void rayGenProgram()
{
    uint32_t seed = tea<64>(theLaunchDim.x * theLaunchIndex.y + theLaunchIndex.x, 0);

    float3 radiance = make_float3(0.0f, 0.0f, 0.0f);
    for (int n = 0; n < numSamples; n++)
    {
        float s = float(theLaunchIndex.x+randf(seed)) / float(theLaunchDim.x);
        float t = float(theLaunchIndex.y+randf(seed)) / float(theLaunchDim.y);

        // generateRay is found in scene/camera.cuh
        optix::Ray theRay = generateRay(s, t, seed);
        float3 sampleRadiance = color(theRay, seed);

        // Remove NaNs - should also remove from sample count? as this is a "bad" sample
        sampleRadiance = removeNaNs(sampleRadiance);

        radiance += sampleRadiance;
    }
    radiance /= numSamples;

    // gamma correction (2)
    radiance = make_float3(
        sqrtf(radiance.x),
        sqrtf(radiance.y),
        sqrtf(radiance.z)
        );

    sysOutputBuffer[theLaunchIndex] = radiance;
}
