#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.cuh"
#include "per_ray_data.cuh"
#include "light_definition.cuh"
#include "shader_common.cuh"

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

rtBuffer<LightDefinition> sysLightDefinitions;

rtDeclareVariable(float, sysEnvironmentRotation, , );


// Not actually a light. Never appears inside the sysLightDefinitions.
RT_PROGRAM void miss_environment_null()
{
  thePrd.radiance = make_float3(0.0f);

#if USE_DENOISER
#if USE_DENOISER_ALBEDO
  thePrd.albedo   = make_float3(0.0f);
#endif
#endif

  thePrd.flags |= FLAG_TERMINATE;  // Not a light.

}

RT_PROGRAM void miss_environment_constant()
{
#if USE_NEXT_EVENT_ESTIMATION
  // If the last surface intersection was a diffuse which was directly lit with multiple importance sampling,
  // then calculate light emission with multiple importance sampling as well.
  const float weightMIS = (thePrd.flags & FLAG_DIFFUSE) ? powerHeuristic(thePrd.pdf, 0.25f * M_1_PIf) : 1.0f;
  thePrd.radiance = make_float3(weightMIS); // Constant white emission multiplied by MIS weight.
#else
  thePrd.radiance = make_float3(1.0f); // Constant white emission.
#endif

#if USE_DENOISER
#if USE_DENOISER_ALBEDO
  thePrd.albedo = make_float3(1.0f); // Constant white emission.
#endif
#endif

  thePrd.flags |= (FLAG_LIGHT | FLAG_TERMINATE);
}

RT_PROGRAM void miss_environment_mapping()
{
  const LightDefinition light = sysLightDefinitions[0];

  const float3 R = theRay.direction;
  // The seam u == 0.0 == 1.0 is in positive z-axis direction.
  // Compensate for the environment rotation done inside the direct lighting.
  const float u     = (atan2f(R.x, -R.z) + M_PIf) * 0.5f * M_1_PIf + sysEnvironmentRotation; // DAR FIXME Use a light.matrix to rotate the environment.
  const float theta = acosf(-R.y);     // theta == 0.0f is south pole, theta == M_PIf is north pole.
  const float v     = theta * M_1_PIf; // Texture is with origin at lower left, v == 0.0f is south pole.

  const float3 emission = make_float3(optix::rtTex2D<float4>(light.idEnvironmentTexture, u, v));

#if USE_NEXT_EVENT_ESTIMATION
  float weightMIS = 1.0f;
  // If the last surface intersection was a diffuse event which was directly lit with multiple importance sampling,
  // then calculate light emission with multiple importance sampling for this implicit light hit as well.
  if (thePrd.flags & FLAG_DIFFUSE)
  {
    // For simplicity we pretend that we perfectly importance-sampled the actual texture-filtered environment map
    // and not the Gaussian smoothed one used to actually generate the CDFs.
    const float pdfLight = intensity(emission) / light.environmentIntegral;
    weightMIS = powerHeuristic(thePrd.pdf, pdfLight);
  }
  thePrd.radiance = emission * weightMIS;
#else
  thePrd.radiance = emission;
#endif

#if USE_DENOISER
#if USE_DENOISER_ALBEDO
  thePrd.albedo = emission;
#endif
#endif

  thePrd.flags |= (FLAG_LIGHT | FLAG_TERMINATE);
}
