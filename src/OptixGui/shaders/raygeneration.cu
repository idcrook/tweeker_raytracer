#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.cuh"
#include "per_ray_data.cuh"
#include "shader_common.cuh"

#include "rt_assert.cuh"

rtBuffer<float4,  2> sysOutputBuffer; // RGBA32F

rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float,    sysSceneEpsilon, , );
rtDeclareVariable(int2,     sysPathLengths, , );
rtDeclareVariable(int,      sysIterationIndex, , );

rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float3, sysCameraU, , );
rtDeclareVariable(float3, sysCameraV, , );
rtDeclareVariable(float3, sysCameraW, , );

rtDeclareVariable(uint2, theLaunchDim,   rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

RT_FUNCTION void integrator(PerRayData& prd, float3& radiance)
{
  radiance = make_float3(0.0f); // Start with black.

  float3 throughput = make_float3(1.0f); // The throughput for the next radiance, starts with 1.0f.

  // Russian Roulette path termination after a specified number of bounces needs the current depth.
  int depth = 0; // Path segment index. Primary ray is 0.

  prd.flags = 0;

  while (depth < sysPathLengths.y)
  {
    prd.wo        = -prd.wi;           // Direction to observer.
    prd.flags    &= FLAG_CLEAR_MASK;   // Clear all non-persistent flags. In this demo only the last diffuse surface interaction stays.

    // Note that the primary rays wouldn't offset the ray t_min by sysSceneEpsilon.
    optix::Ray ray = optix::make_Ray(prd.pos, prd.wi, 0, sysSceneEpsilon, RT_DEFAULT_MAX);
    rtTrace(sysTopObject, ray, prd);

    radiance += throughput * prd.radiance;

    // Path termination by miss shader or sample() routines.
    // If terminate is true, f_over_pdf and pdf might be undefined.
    if ((prd.flags & FLAG_TERMINATE) || prd.pdf <= 0.0f || isNull(prd.f_over_pdf))
    {
      break;
    }

    // PERF f_over_pdf already contains the proper throughput adjustment for diffuse materials: f * (fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf);
    throughput *= prd.f_over_pdf;

    // Unbiased Russian Roulette path termination.
    if (sysPathLengths.x <= depth) // Start termination after a minimum number of bounces.
    {
      const float probability = fmaxf(throughput); // DAR Other options: // intensity(throughput); // fminf(0.5f, intensity(throughput));
      if (probability < rng(prd.seed)) // Paths with lower probability to continue are terminated earlier.
      {
        break;
      }
      throughput /= probability; // Path isn't terminated. Adjust the throughput so that the average is right again.
    }

    ++depth; // Next path segment.
  }
}

// Entry point for pinhole camera with manual accumulation, non-VCA.
RT_PROGRAM void raygeneration()
{
  PerRayData prd;

  // Initialize the random number generator seed from the linear pixel index and the iteration index.
  prd.seed = tea<8>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, sysIterationIndex);

  // Pinhole camera implementation:
  // The launch index is the pixel coordinate.
  // Note that launchIndex = (0, 0) is the bottom left corner of the image,
  // which matches the origin in the OpenGL texture used to display the result.
  const float2 pixel = make_float2(theLaunchIndex);
  // Sample the ray in the center of the pixel.
  const float2 fragment = pixel + rng2(prd.seed); // Random jitter of the fragment location in this pixel.
  // The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
  const float2 screen = make_float2(theLaunchDim);
  // Normalized device coordinates in range [-1, 1].
  const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

  // The integrator expects the next path segments ray.origin in prd.pos and the next ray.direction in prd.wi.
  prd.pos = sysCameraPosition;
  prd.wi  = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

  float3 radiance;

  integrator(prd, radiance); // In this case a unidirectional path tracer.

#if USE_DEBUG_EXCEPTIONS
  // DAR DEBUG Highlight numerical errors.
  if (isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z))
  {
    radiance = make_float3(1000000.0f, 0.0f, 0.0f); // super red
  }
  else if (isinf(radiance.x) || isinf(radiance.y) || isinf(radiance.z))
  {
    radiance = make_float3(0.0f, 1000000.0f, 0.0f); // super green
  }
  else if (radiance.x < 0.0f || radiance.y < 0.0f || radiance.z < 0.0f)
  {
    radiance = make_float3(0.0f, 0.0f, 1000000.0f); // super blue
  }
#else
  // NaN values will never go away. Filter them out before they can arrive in the output buffer.
  // This only has an effect if the debug coloring above is off!
  if (!(isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z)))
#endif
  {
    if (0 < sysIterationIndex)
    {
      float4 dst = sysOutputBuffer[theLaunchIndex];  // RGBA32F
      sysOutputBuffer[theLaunchIndex] = optix::lerp(dst, make_float4(radiance, 1.0f), 1.0f / (float) (sysIterationIndex + 1));
    }
    else
    {
      // sysIterationIndex 0 will fill the buffer.
      // If this isn't done separately, the result of the lerp() above is undefined, e.g. dst could be NaN.
      sysOutputBuffer[theLaunchIndex] = make_float4(radiance, 1.0f);
    }
  }
}
