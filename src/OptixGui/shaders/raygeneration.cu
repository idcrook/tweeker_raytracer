#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.cuh"
#include "per_ray_data.cuh"
#include "shader_common.cuh"

#include "rt_assert.cuh"

rtBuffer<float4, 2> sysOutputBuffer; // RGBA32F

#if USE_DENOISER
rtBuffer<float4, 2> sysAlbedoBuffer; // RGBA32F
#endif

rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float,    sysSceneEpsilon, , );
rtDeclareVariable(int2,     sysPathLengths, , );
rtDeclareVariable(int,      sysIterationIndex, , );
rtDeclareVariable(int,      sysCameraType, , );
rtDeclareVariable(int,      sysShutterType, , );

// Bindless callable programs implementing different lens shaders.
rtBuffer< rtCallableProgramId<void(const float2 pixel, const float2 screen, const float2 sample, float3& origin, float3& direction)> > sysLensShader;

rtDeclareVariable(uint2, theLaunchDim,   rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

RT_FUNCTION void integrator(PerRayData& prd, float3& radiance, float3& albedo)
{
  // This renderer supports nested volumes. Four levels is plenty enough for most cases.
  // The absorption coefficient and IOR of the volume the ray is currently inside.
  float4 absorptionStack[MATERIAL_STACK_SIZE]; // .xyz == absorptionCoefficient (sigma_a), .w == index of refraction

  radiance = make_float3(0.0f); // Start with black.
  albedo   = make_float3(0.0f); // Start with black.

  // case 0: Standard stochastic motion blur.
  float time = rng(prd.seed); // Set the time of this path to a random value in the range [0, 1).

  switch (sysShutterType) // In case another camera shutter is active reuse that random value.
  {
    case 1: // Rolling shutter from top to bottom.
      // Note that launchIndex (0, 0) is as the bottom left corner, which matches what OpenGL expects as texture orientation.
      // Each row gets a different time plus some stochastic antialiasing on that line.
      time = (float(theLaunchDim.y - 1 - theLaunchIndex.y) + time) / float(theLaunchDim.y);
      break;
    case 2: // Rolling shutter from bottom to top.
      time = (float(theLaunchIndex.y) + time) / float(theLaunchDim.y);
      break;
    case 3: // Rolling shutter from left to right.
      time = (float(theLaunchIndex.x) + time) / float(theLaunchDim.x);
      break;
    case 4: // Rolling shutter from right to left.
      time = (float(theLaunchDim.x - 1 - theLaunchIndex.x) + time) / float(theLaunchDim.x);
      break;
  }

  float3 throughput = make_float3(1.0f); // The throughput for the next radiance, starts with 1.0f.

  int stackIdx = MATERIAL_STACK_EMPTY; // Start with empty nested materials stack.
  int depth = 0;                       // Path segment index. Primary ray is 0.

  prd.absorption_ior = make_float4(0.0f, 0.0f, 0.0f, 1.0f); // Assume primary ray starts in vacuum.

  prd.flags = 0;

  // Russian Roulette path termination after a specified number of bounces needs the current depth.
  while (depth < sysPathLengths.y)
  {
    prd.wo        = -prd.wi;           // Direction to observer.
    prd.ior       = make_float2(1.0f); // Reset the volume IORs.
    prd.distance  = RT_DEFAULT_MAX;    // Shoot the next ray with maximum length.
    prd.flags    &= FLAG_CLEAR_MASK;   // Clear all non-persistent flags. In this demo only the last diffuse surface interaction stays.

    // Handle volume absorption of nested materials.
    if (MATERIAL_STACK_FIRST <= stackIdx) // Inside a volume?
    {
      prd.flags     |= FLAG_VOLUME;                            // Indicate that we're inside a volume. => At least absorption calculation needs to happen.
      prd.extinction = make_float3(absorptionStack[stackIdx]); // There is only volume absorption in this demo, no volume scattering.
      prd.ior.x      = absorptionStack[stackIdx].w;            // The IOR of the volume we're inside. Needed for eta calculations in transparent materials.
      if (MATERIAL_STACK_FIRST <= stackIdx - 1)
      {
        prd.ior.y = absorptionStack[stackIdx - 1].w; // The IOR of the surrounding volume. Needed when potentially leaving a volume to calculate eta in transparent materials.
      }
    }

    // Note that the primary rays (or volume scattering miss cases) wouldn't normally offset the ray t_min by sysSceneEpsilon. Keep it simple here.
    optix::Ray ray = optix::make_Ray(prd.pos, prd.wi, 0, sysSceneEpsilon, prd.distance);
    // Note that this time defines the semantic variable rtCurrentTime in the other program domains.
    rtTrace(sysTopObject, ray, time, prd);

    // This renderer supports nested volumes.
    if (prd.flags & FLAG_VOLUME)
    {
      // We're inside a volume. Calculate the extinction along the current path segment in any case.
      // The transmittance along the current path segment inside a volume needs to attenuate the ray throughput with the extinction
      // before it modulates the radiance of the hitpoint.
      throughput *= expf(-prd.distance * prd.extinction);
    }

    radiance += throughput * prd.radiance;

#if USE_DENOISER
    // In physical terms, the albedo is a single color value approximating the ratio of radiant exitance to the irradiance under uniform lighting.
    // The albedo value can be approximated for simple materials by using the diffuse color of the first hit,
    // or for layered materials by using a weighted sum of the individual BRDFs albedo values.
    // For some objects such as perfect mirrors, the quality of the result might be improved by using the albedo value of a subsequent hit instead.
#if 1
    // When no albedo has been written before and the hit was diffuse or a light, write the albedo.
    // DAR This makes glass materials and motion blur on specular surfaces in the demo a little noisier,
    // but should definitely be used with high frequency textures behind transparent or around reflective materials.
    if (!(prd.flags & FLAG_ALBEDO) && (prd.flags & (FLAG_DIFFUSE | FLAG_LIGHT)))
    {
      // The albedo buffer should contain the surface appearance under uniform lighting in linear color space in the range [0.0f, 1.0f].
      // Clamp the final albedo result to that range here, because it captured the radiance when hitting lights either directly or via specular events.
      albedo = optix::clamp(throughput * prd.albedo, 0.0f, 1.0f);
      prd.flags |= FLAG_ALBEDO; // This flag is persistent along the path and prevents that the albedo is written more than once.
    }
#else
    if (depth == 0) // Just write the albedo of the primary ray.
    {
      albedo = optix::clamp(throughput * prd.albedo, 0.0f, 1.0f); // See comment above. Expects linear colors in the range [0.0f, 1.0f]
    }
#endif

#endif // USE_DENOISER

    // Path termination by miss shader or sample() routines.
    // If terminate is true, f_over_pdf and pdf might be undefined.
    if ((prd.flags & FLAG_TERMINATE) || prd.pdf <= 0.0f || isNull(prd.f_over_pdf))
    {
      break;
    }

    // PERF f_over_pdf already contains the proper throughput adjustment for diffuse materials: f * (fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf);
    throughput *= prd.f_over_pdf;

    // Unbiased Russian Roulette path termination.
    if (sysPathLengths.x <= depth) // Start termination after a minimum number of bounces.
    {
      const float probability = fmaxf(throughput); // DAR Other options: // intensity(throughput); // fminf(0.5f, intensity(throughput));
      if (probability < rng(prd.seed)) // Paths with lower probability to continue are terminated earlier.
      {
        break;
      }
      throughput /= probability; // Path isn't terminated. Adjust the throughput so that the average is right again.
    }

    // Adjust the material volume stack if the geometry is not thin-walled but a border between two volumes
    // and the outgoing ray direction was a transmission.
    if ((prd.flags & (FLAG_THINWALLED | FLAG_TRANSMISSION)) == FLAG_TRANSMISSION)
    {
      // Transmission.
      if (prd.flags & FLAG_FRONTFACE) // Entered a new volume?
      {
        // Push the entered material's volume properties onto the volume stack.
        //rtAssert((stackIdx < MATERIAL_STACK_LAST), 1); // Overflow?
        stackIdx = min(stackIdx + 1, MATERIAL_STACK_LAST);
        absorptionStack[stackIdx] = prd.absorption_ior;
      }
      else // Exited the current volume?
      {
        // Pop the top of stack material volume.
        // This assert fires and is intended because I tuned the frontface checks so that there are more exits than enters at silhouettes.
        //rtAssert((MATERIAL_STACK_EMPTY < stackIdx), 0); // Underflow?
        stackIdx = max(stackIdx - 1, MATERIAL_STACK_EMPTY);
      }
    }

    ++depth; // Next path segment.
  }
}

RT_PROGRAM void raygeneration()
{
  PerRayData prd;

  // Initialize the random number generator seed from the linear pixel index and the iteration index.
  prd.seed = tea<8>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, sysIterationIndex);

  // DAR Decoupling the pixel coordinates from the screen size will allow for partial rendering algorithms.
  // In this case theLaunchIndex is the pixel coordinate and theLaunchDim is sysOutputBuffer.size().
  sysLensShader[sysCameraType](make_float2(theLaunchIndex), make_float2(theLaunchDim), rng2(prd.seed), prd.pos, prd.wi); // Calculate the primary ray with a lens shader program.

  float3 radiance;
  float3 albedo;

  integrator(prd, radiance, albedo); // In this case a unidirectional path tracer.

#if USE_DEBUG_EXCEPTIONS
  // DAR DEBUG Highlight numerical errors.
  if (isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z))
  {
    radiance = make_float3(1000000.0f, 0.0f, 0.0f); // super red
  }
  else if (isinf(radiance.x) || isinf(radiance.y) || isinf(radiance.z))
  {
    radiance = make_float3(0.0f, 1000000.0f, 0.0f); // super green
  }
  else if (radiance.x < 0.0f || radiance.y < 0.0f || radiance.z < 0.0f)
  {
    radiance = make_float3(0.0f, 0.0f, 1000000.0f); // super blue
  }
#else
  // NaN values will never go away. Filter them out before they can arrive in the output buffer.
  // This only has an effect if the debug coloring above is off!
  if (!(isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z)))
#endif
  {
    if (0 < sysIterationIndex)
    {
      float4 dst = sysOutputBuffer[theLaunchIndex];  // RGBA32F
      sysOutputBuffer[theLaunchIndex] = optix::lerp(dst, make_float4(radiance, 1.0f), 1.0f / (float) (sysIterationIndex + 1));

#if USE_DENOISER
      dst = sysAlbedoBuffer[theLaunchIndex];  // RGBA32F
      sysAlbedoBuffer[theLaunchIndex] = optix::lerp(dst, make_float4(albedo, 1.0f), 1.0f / (float) (sysIterationIndex + 1));
#endif

    }
    else
    {
      // sysIterationIndex 0 will fill the buffer.
      // If this isn't done separately, the result of the lerp() above is undefined, e.g. dst could be NaN.
      sysOutputBuffer[theLaunchIndex] = make_float4(radiance, 1.0f);

#if USE_DENOISER
      sysAlbedoBuffer[theLaunchIndex] = make_float4(albedo, 1.0f);
#endif

    }
  }
}

#if USE_DENOISER
// OptiX 5.0.x needs at least one appendLaunch() in the post-processing CommandList or the denoiser will not trigger its memory allocations.
// This is fixed in OptiX 5.1.0 which also supports HDR denoising directly, so that the tonemapper can be placed last again.
// Put my own tonemapper from GLSL here to get the proper gamma corrected input into the denoiser.

rtBuffer<float4, 2> sysTonemappedBuffer;

rtDeclareVariable(float3, sysColorBalance, , );
rtDeclareVariable(float,  sysInvGamma, , );
rtDeclareVariable(float,  sysInvWhitePoint, , );
rtDeclareVariable(float,  sysBurnHighlights, , );
rtDeclareVariable(float,  sysCrushBlacks, , );
rtDeclareVariable(float,  sysSaturation, , );

RT_PROGRAM void raygeneration_tonemapper()
{
  const float3 hdrColor = make_float3(sysOutputBuffer[theLaunchIndex]);

  float3 ldrColor = sysInvWhitePoint * sysColorBalance * hdrColor;
  ldrColor *= (ldrColor * make_float3(sysBurnHighlights) + make_float3(1.0f)) / (ldrColor + make_float3(1.0f));

  float luminance = optix::dot(ldrColor, make_float3(0.3f, 0.59f, 0.11f));
  ldrColor = optix::lerp(make_float3(luminance), ldrColor, sysSaturation); // This can generate negative values for sysSaturation > 1.0f!
  ldrColor = fmaxf(0.0f, ldrColor); // Prevent negative values.

  luminance = optix::dot(ldrColor, make_float3(0.3f, 0.59f, 0.11f));
  if (luminance < 1.0f)
  {
    const float3 crushed = powf(ldrColor, sysCrushBlacks);
    ldrColor = optix::lerp(crushed, ldrColor, sqrtf(luminance));
    ldrColor = fmaxf(0.0f, ldrColor); // Prevent negative values.
  }
  ldrColor = powf(ldrColor, sysInvGamma);

#if USE_DEBUG_EXCEPTIONS
  // DAR DEBUG Highlight numerical errors.
  if (isnan(ldrColor.x) || isnan(ldrColor.y) || isnan(ldrColor.z))
  {
    ldrColor = make_float3(1000000.0f, 0.0f, 0.0f); // super red
  }
  else if (isinf(ldrColor.x) || isinf(ldrColor.y) || isinf(ldrColor.z))
  {
    ldrColor = make_float3(0.0f, 1000000.0f, 0.0f); // super green
  }
  else if (ldrColor.x < 0.0f || ldrColor.y < 0.0f || ldrColor.z < 0.0f)
  {
    ldrColor = make_float3(0.0f, 0.0f, 1000000.0f); // super blue
  }
#endif

  sysTonemappedBuffer[theLaunchIndex] = make_float4(ldrColor, 1.0f);
}
#endif // USE_DENOISER
