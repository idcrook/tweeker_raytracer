#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.cuh"
#include "per_ray_data.cuh"

#include "rt_assert.cuh"

rtBuffer<float4,  2> sysOutputBuffer; // RGBA32F

rtDeclareVariable(rtObject, sysTopObject, , );

rtDeclareVariable(uint2, theLaunchDim,   rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float3, sysCameraU, , );
rtDeclareVariable(float3, sysCameraV, , );
rtDeclareVariable(float3, sysCameraW, , );


// Entry point for a pinhole camera.
RT_PROGRAM void raygeneration()
{
  PerRayData prd;

  prd.radiance = make_float3(0.0f);

  // The launch index is the pixel coordinate.
  // Note that launchIndex = (0, 0) is the bottom left corner of the image,
  // which matches the origin in the OpenGL texture used to display the result.
  const float2 pixel = make_float2(theLaunchIndex);
  // Sample the ray in the center of the pixel.
  const float2 fragment = pixel + make_float2(0.5f);
  // The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
  const float2 screen = make_float2(theLaunchDim);
  // Normalized device coordinates in range [-1, 1].
  const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

  const float3 origin    = sysCameraPosition;
  const float3 direction = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

  // Shoot a ray from origin into direction (must always be normalized!) for ray type 0 and test the interval between 0.0 and RT_DEFAULT_MAX for intersections.
  // There is no geometry in the scene, yet, so this will always invoke the miss program assigned to ray type 0, which is the radiance ray in this implementation.
  optix::Ray ray = optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);

  // Start the ray traversal at the scene's root node, which in this case is an empty Group.
  // The ray becomes the variable with rtCurrentRay semantic in the other program domains.
  // The PerRayData becomes the variable with the semantic rtPayload in the other program domains,
  // which allows to exchange arbitrary data between the program domains.
  rtTrace(sysTopObject, ray, prd);

  sysOutputBuffer[theLaunchIndex] = make_float4(prd.radiance, 1.0f);
}
