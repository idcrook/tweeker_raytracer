#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.cuh"
#include "per_ray_data.cuh"
#include "material_parameter.cuh"
#include "light_definition.cuh"
#include "shader_common.cuh"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float,    sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay,                  rtCurrentRay, );
rtDeclareVariable(float,      theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd,                  rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
//rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

// Material parameter definition.
rtBuffer<MaterialParameter> sysMaterialParameters; // Context global buffer with an array of structures of MaterialParameter.
rtDeclareVariable(int,      parMaterialIndex, , ); // Per Material index into the sysMaterialParameters array.

rtBuffer<LightDefinition> sysLightDefinitions;
rtDeclareVariable(int,    sysNumLights, , );     // PERF Used many times and faster to read than sysLightDefinitions.size().

rtBuffer< rtCallableProgramId<void(float3 const& point, const float2 sample, LightSample& lightSample)> > sysSampleLight;


// Helper functions for sampling a cosine weighted hemisphere distrobution as needed for the Lambert shading model.

RT_FUNCTION void alignVector(float3 const& axis, float3& w)
{
  // Align w with axis.
  const float s = copysign(1.0f, axis.z);
  w.z *= s;
  const float3 h = make_float3(axis.x, axis.y, axis.z + s);
  const float  k = optix::dot(w, h) / (1.0f + fabsf(axis.z));
  w = k * h - w;
}

RT_FUNCTION void unitSquareToCosineHemisphere(const float2 sample, float3 const& axis, float3& w, float& pdf)
{
  // Choose a point on the local hemisphere coordinates about +z.
  const float theta = 2.0f * M_PIf * sample.x;
  const float r = sqrtf(sample.y);
  w.x = r * cosf(theta);
  w.y = r * sinf(theta);
  w.z = 1.0f - w.x * w.x - w.y * w.y;
  w.z = (0.0f < w.z) ? sqrtf(w.z) : 0.0f;

  pdf = w.z * M_1_PIf;

  // Align with axis.
  alignVector(axis, w);
}

RT_PROGRAM void closesthit()
{
  float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
  float3 normal    = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

  thePrd.pos = theRay.origin + theRay.direction * theIntersectionDistance; // Advance the path to the hit position in world coordinates.

  // Explicitly include edge-on cases as frontface condition! (Important for nested materials shown in a later example.)
  thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, geoNormal)) ? FLAG_FRONTFACE : 0;

  if ((thePrd.flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
  {
    // Means geometric normal and shading normal are always defined on the side currently looked at.
    // This gives the backfaces of opaque BSDFs a defined result.
    geoNormal = -geoNormal;
    normal    = -normal;
    // Do not recalculate the frontface condition!
  }

  // A material system with support for arbitrary mesh lights would evaluate its emission here.
  // But since only parallelogram area lights are supported, those get a dedicated closest hit program to simplify this demo.
  thePrd.radiance = make_float3(0.0f);

  // Start fresh with the next BSDF sample.  (Either of these values remaining zero is an end-of-path condition.)
  thePrd.f_over_pdf = make_float3(0.0f);
  thePrd.pdf        = 0.0f;

  // Lambert sampling:
  // Cosine weighted hemisphere sampling above the shading normal.
  // This calculates the ray.direction for the next path segment in wi and its probability density function value in pdf.
  unitSquareToCosineHemisphere(rng2(thePrd.seed), normal, thePrd.wi, thePrd.pdf);

  // Do not sample opaque surfaces below the geometry!
  // Mind that the geometry normal has been flipped to the side the ray points at.
  if (thePrd.pdf <= 0.0f || optix::dot(thePrd.wi, geoNormal) <= 0.0f)
  {
    thePrd.flags |= FLAG_TERMINATE;
    return;
  }

  MaterialParameter parameters = sysMaterialParameters[parMaterialIndex];

  thePrd.f_over_pdf = parameters.albedo * (M_1_PIf * optix::dot(thePrd.wi, normal) / thePrd.pdf); // PERF wi and normal are in the same hemisphere, no fabsf() needed on the cosTheta.
  thePrd.flags     |= FLAG_DIFFUSE;

#if USE_NEXT_EVENT_ESTIMATION
  // Direct lighting if the sampled BSDF was diffuse and any light is in the scene.
  if ( /* (thePrd.flags & FLAG_DIFFUSE) && */ 0 < sysNumLights) // No need to check FLAG_DIFFUSE. That has been set one line above. See in optixIntro_07 when this is needed.
  {
    const float2 sample = rng2(thePrd.seed); // Use lower dimension samples for the position. (Irrelevant for the LCG).

    LightSample lightSample; // Sample one of many lights.

    // The caller picks the light to sample. Make sure the index stays in the bounds of the sysLightDefinitions array.
    lightSample.index = optix::clamp(static_cast<int>(floorf(rng(thePrd.seed) * sysNumLights)), 0, sysNumLights - 1);

    const LightType lightType = sysLightDefinitions[lightSample.index].type;

    sysSampleLight[lightType](thePrd.pos, sample, lightSample); // lightSample direction and distance returned in world space!

    if (0.0f < lightSample.pdf) // Useful light sample?
    {
      // Lambert evaluation
      // Evaluate the Lambert BSDF in the light sample direction. Normally cheaper than shooting rays.
      const float3 f   = parameters.albedo * M_1_PIf;
      const float  pdf = fmaxf(0.0f, optix::dot(lightSample.direction, normal) * M_1_PIf);

      if (0.0f < pdf && isNotNull(f))
      {
        // Do the visibility check of the light sample.
        PerRayData_shadow prdShadow;

        prdShadow.visible = true; // Initialize for miss.

        // Note that the sysSceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval
        // to prevent self intersections with the actual light geometry in the scene!
        optix::Ray ray = optix::make_Ray(thePrd.pos, lightSample.direction, 1, sysSceneEpsilon, lightSample.distance - sysSceneEpsilon); // Shadow ray.
        rtTrace(sysTopObject, ray, prdShadow);

        if (prdShadow.visible)
        {
          const float misWeight = powerHeuristic(lightSample.pdf, pdf);

          thePrd.radiance += f * lightSample.emission * (misWeight * optix::dot(lightSample.direction, normal) / lightSample.pdf);
        }
      }
    }
  }
#endif // USE_NEXT_EVENT_ESTIMATION
}
