#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.cuh"
#include "per_ray_data.cuh"
#include "material_parameter.cuh"
#include "light_definition.cuh"
#include "shader_common.cuh"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float,    sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay,                  rtCurrentRay, );
rtDeclareVariable(float,      theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd,                  rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

// Material parameter definition.
rtBuffer<MaterialParameter> sysMaterialParameters; // Context global buffer with an array of structures of MaterialParameter.
rtDeclareVariable(int,      parMaterialIndex, , ); // Per Material index into the sysMaterialParameters array.

rtBuffer<LightDefinition> sysLightDefinitions;
rtDeclareVariable(int,    sysNumLights, , );     // PERF Used many times and faster to read than sysLightDefinitions.size().

rtBuffer< rtCallableProgramId<void(MaterialParameter const& parameters, State const& state, PerRayData& prd)> > sysSampleBSDF;
rtBuffer< rtCallableProgramId<float4(MaterialParameter const& parameters, State const& state, PerRayData const& prd, float3 const& wiL)> > sysEvalBSDF;

rtBuffer< rtCallableProgramId<void(float3 const& point, const float2 sample, LightSample& lightSample)> > sysSampleLight;

RT_PROGRAM void closesthit()
{
  State state; // All in world space coordinates!

  state.geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
  state.normal    = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));
  state.texcoord  = varTexCoord;

  thePrd.pos      = theRay.origin + theRay.direction * theIntersectionDistance; // Advance the path to the hit position in world coordinates.
  thePrd.distance = theIntersectionDistance; // Return the current path segment distance, needed for absorption calculations in the integrator.

  // Explicitly include edge-on cases as frontface condition!
  // Keeps the material stack from overflowing at silhouettes.
  // Prevents that silhouettes of thin-walled materials use the backface material.
  // Using the true geometry normal attribute as originally defined on the frontface!
  thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, state.geoNormal)) ? FLAG_FRONTFACE : 0;

  if ((thePrd.flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
  {
    // Means geometric normal and shading normal are always defined on the side currently looked at.
    // This gives the backfaces of opaque BSDFs a defined result.
    state.geoNormal = -state.geoNormal;
    state.normal    = -state.normal;
    // Do not recalculate the frontface condition!
  }

  // A material system with support for arbitrary mesh lights would evaluate its emission here.
  // But since only parallelogram area lights are supported, those get a dedicated closest hit program to simplify this demo.
  thePrd.radiance = make_float3(0.0f);

  MaterialParameter parameters = sysMaterialParameters[parMaterialIndex]; // Copy the material parameters locally to be able to fetch texture data once.

  if (parameters.albedoID != RT_TEXTURE_ID_NULL)
  {
    const float3 texColor = make_float3(optix::rtTex2D<float4>(parameters.albedoID, state.texcoord.x, state.texcoord.y));

    // Modulate the incoming color with the texture.
    parameters.albedo *= texColor;               // linear color, resp. if the texture has been uint8 and readmode set to use sRGB, then sRGB.
    //parameters.albedo *= powf(texColor, 2.2f); // sRGB gamma correction done manually.
  }

  // Start fresh with the next BSDF sample.  (Either of these values remaining zero is an end-of-path condition.)
  thePrd.f_over_pdf = make_float3(0.0f);
  thePrd.pdf        = 0.0f;

  // Only the last diffuse hit is tracked for multiple importance sampling of implicit light hits.
  thePrd.flags = (thePrd.flags & ~FLAG_DIFFUSE) | parameters.flags; // FLAG_THINWALLED can be set directly from the material parameters.

  sysSampleBSDF[parameters.indexBSDF](parameters, state, thePrd);

#if USE_NEXT_EVENT_ESTIMATION
  // Direct lighting if the sampled BSDF was diffuse and any light is in the scene.
  if ((thePrd.flags & FLAG_DIFFUSE) && 0 < sysNumLights)
  {
    const float2 sample = rng2(thePrd.seed); // Use lower dimension samples for the position. (Irrelevant for the LCG).

    LightSample lightSample; // Sample one of many lights.

    // The caller picks the light to sample. Make sure the index stays in the bounds of the sysLightDefinitions array.
    lightSample.index = optix::clamp(static_cast<int>(floorf(rng(thePrd.seed) * sysNumLights)), 0, sysNumLights - 1);

    const LightType lightType = sysLightDefinitions[lightSample.index].type;

    sysSampleLight[lightType](thePrd.pos, sample, lightSample);

    if (0.0f < lightSample.pdf) // Useful light sample?
    {
      // Evaluate the BSDF in the light sample direction. Normally cheaper than shooting rays.
      // Returns BSDF f in .xyz and the BSDF pdf in .w
      const float4 bsdf_pdf = sysEvalBSDF[parameters.indexBSDF](parameters, state, thePrd, lightSample.direction);

      if (0.0f < bsdf_pdf.w && isNotNull(make_float3(bsdf_pdf)))
      {
        // Do the visibility check of the light sample.
        PerRayData_shadow prdShadow;

        prdShadow.seed    = thePrd.seed; // For potential stochastic cutout opacity sampling.
        prdShadow.visible = true;        // Initialize for miss.

        // Note that the sysSceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval
        // to prevent self intersections with the actual light geometry in the scene!
        optix::Ray ray = optix::make_Ray(thePrd.pos, lightSample.direction, 1, sysSceneEpsilon, lightSample.distance - sysSceneEpsilon); // Shadow ray.
        rtTrace(sysTopObject, ray, prdShadow);

        thePrd.seed = prdShadow.seed; // Continue the RNG state!

        if (prdShadow.visible)
        {
          if (thePrd.flags & FLAG_VOLUME) // Supporting nested materials includes having lights inside a volume.
          {
            // Calculate the transmittance along the light sample's distance in case it's inside a volume.
            // The light must be in the same volume or it would have been shadowed!
            lightSample.emission *= expf(-lightSample.distance * thePrd.extinction);
          }

          const float misWeight = powerHeuristic(lightSample.pdf, bsdf_pdf.w);

          thePrd.radiance += make_float3(bsdf_pdf) * lightSample.emission * (misWeight * optix::dot(lightSample.direction, state.normal) / lightSample.pdf);
        }
      }
    }
  }
#endif // USE_NEXT_EVENT_ESTIMATION
}
