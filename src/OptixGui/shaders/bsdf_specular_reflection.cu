#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.cuh"
#include "material_parameter.cuh"
#include "per_ray_data.cuh"

RT_CALLABLE_PROGRAM void sample_bsdf_specular_reflection(MaterialParameter const& parameters, State const& state, PerRayData& prd)
{
  prd.wi = optix::reflect(-prd.wo, state.normal);

  if (optix::dot(prd.wi, state.geoNormal) <= 0.0f) // Do not sample opaque materials below the geometric surface.
  {
    prd.flags |= FLAG_TERMINATE;
    return;
  }

  prd.f_over_pdf = parameters.albedo;
  prd.pdf        = 1.0f;
}

// This is actually never reached, because the FLAG_DIFFUSE flag is not set when a specular BSDF is has been sampled.
RT_CALLABLE_PROGRAM float4 eval_bsdf_specular_reflection(MaterialParameter const& parameters, State const& state, PerRayData const& prd, float3 const& wiL)
{
  return make_float4(0.0f);
}
