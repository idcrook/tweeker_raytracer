#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.cuh"

#include "per_ray_data.cuh"
#include "light_definition.cuh"
#include "shader_common.cuh"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
//rtDeclareVariable(float,    sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay,                  rtCurrentRay, );
rtDeclareVariable(float,      theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd,                  rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
//rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
//rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

rtBuffer<LightDefinition> sysLightDefinitions;
rtDeclareVariable(int,    parLightIndex, , );  // Index into the sysLightDefinitions array.

// Very simple closest hit program just for rectangle area lights.
RT_PROGRAM void closesthit_light()
{
  thePrd.pos      = theRay.origin + theRay.direction * theIntersectionDistance; // Advance the path to the hit position in world coordinates.
  thePrd.distance = theIntersectionDistance; // Return the current path segment distance, needed for absorption calculations in the integrator.

  const float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal)); // PERF Not really needed when it's know that light geometry is not under Transforms.

  const float cosTheta = optix::dot(thePrd.wo, geoNormal);
  thePrd.flags |= (0.0f <= cosTheta) ? FLAG_FRONTFACE : 0;

  thePrd.radiance = make_float3(0.0f); // Backside is black.

  if (thePrd.flags & FLAG_FRONTFACE) // Looking at the front face?
  {
    const LightDefinition light = sysLightDefinitions[parLightIndex];

    thePrd.radiance = light.emission;

#if USE_NEXT_EVENT_ESTIMATION
    const float pdfLight = (thePrd.distance * thePrd.distance) / (light.area * cosTheta); // Solid angle pdf. Assumes light.area != 0.0f.
    // If it's an implicit light hit from a diffuse scattering event and the light emission was not returning a zero pdf.
    if ((thePrd.flags & FLAG_DIFFUSE) && DENOMINATOR_EPSILON < pdfLight)
    {
      // Scale the emission with the power heuristic between the previous BSDF sample pdf and this implicit light sample pdf.
      thePrd.radiance *= powerHeuristic(thePrd.pdf, pdfLight);
    }
#endif // USE_NEXT_EVENT_ESTIMATION
  }

  // Lights have no other material properties than emission in this demo. Terminate the path.
  thePrd.flags |= FLAG_TERMINATE;
}
