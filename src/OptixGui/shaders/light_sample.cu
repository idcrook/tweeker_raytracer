#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.cuh"
#include "per_ray_data.cuh"
#include "light_definition.cuh"
#include "shader_common.cuh"

#include "rt_assert.cuh"

rtBuffer<LightDefinition> sysLightDefinitions;
rtDeclareVariable(int,    sysNumLights, , ); // PERF Used many times and faster to read than sysLightDefinitions.size().

rtDeclareVariable(float,  sysEnvironmentRotation, , );


RT_FUNCTION void unitSquareToSphere(const float u, const float v, float3& p, float& pdf)
{
  p.z = 1.0f - 2.0f * u;
  float r = 1.0f - p.z * p.z;
  r = (0.0f < r) ? sqrtf(r) : 0.0f;

  const float phi = v * 2.0f * M_PIf;
  p.x = r * cosf(phi);
  p.y = r * sinf(phi);

  pdf = 0.25f * M_1_PIf;  // == 1.0f / (4.0f * M_PIf)
}

// Note that all light sampling routines return lightSample.direction and lightSample.distance in world space!

RT_CALLABLE_PROGRAM void sample_light_constant(float3 const& point, const float2 sample, LightSample& lightSample)
{
  unitSquareToSphere(sample.x, sample.y, lightSample.direction, lightSample.pdf);

  // Environment lights do not set the light sample position!
  lightSample.distance = RT_DEFAULT_MAX; // Environment light.

  // Explicit light sample. White scaled by inverse probabilty to hit this light.
  lightSample.emission = make_float3(sysNumLights);
}

RT_CALLABLE_PROGRAM void sample_light_environment(float3 const& point, const float2 sample, LightSample& lightSample)
{
  const LightDefinition light = sysLightDefinitions[0]; // The environment light is always placed into the first entry.

  // Importance-sample the spherical environment light direction.
  const unsigned int sizeU = static_cast<unsigned int>(light.idEnvironmentCDF_U.size().x);
  const unsigned int sizeV = static_cast<unsigned int>(light.idEnvironmentCDF_V.size());

  unsigned int ilo = 0;          // Use this for full spherical lighting. (This matches the result of indirect environment lighting.)
  unsigned int ihi = sizeV - 1 ; // Index on the last entry containing 1.0f. Can never be reached with the sample in the range [0.0f, 1.0f).

  // Binary search the row index to look up.
  while (ilo != ihi - 1) // When a pair of limits have been found, the lower index indicates the cell to use.
  {
    const unsigned int i = (ilo + ihi) >> 1;
    const float cdf = light.idEnvironmentCDF_V[i];
    if (sample.y < cdf) // If the cdf is greater than the sample, use that as new higher limit.
    {
      ihi = i;
    }
    else // If the sample is greater than or equal to the CDF value, use that as new lower limit.
    {
      ilo = i;
    }
  }

  uint2 index; // 2D index used in the next binary search.
  index.y = ilo; // This is the row we found.

  // Binary search the column index to look up.
  ilo = 0;
  ihi = sizeU - 1; // Index on the last entry containing 1.0f. Can never be reached with the sample in the range [0.0f, 1.0f).
  while (ilo != ihi - 1) // When a pair of limits have been found, the lower index indicates the cell to use.
  {
    index.x = (ilo + ihi) >> 1;
    const float cdf = light.idEnvironmentCDF_U[index];
    if (sample.x < cdf) // If the CDF value is greater than the sample, use that as new higher limit.
    {
      ihi = index.x;
    }
    else // If the sample is greater than or equal to the CDF value, use that as new lower limit.
    {
      ilo = index.x;
    }
  }

  index.x = ilo; // The column result.

  // Continuous sampling of the CDF.
  const float cdfLowerU = light.idEnvironmentCDF_U[index];
  const float cdfUpperU = light.idEnvironmentCDF_U[make_uint2(index.x + 1, index.y)];
  const float du = (sample.x - cdfLowerU) / (cdfUpperU - cdfLowerU);

  const float cdfLowerV = light.idEnvironmentCDF_V[index.y];
  const float cdfUpperV = light.idEnvironmentCDF_V[index.y + 1];
  const float dv = (sample.y - cdfLowerV) / (cdfUpperV - cdfLowerV);

  // Texture lookup coordinates.
  const float u = (float(index.x) + du) / float(sizeU - 1);
  const float v = (float(index.y) + dv) / float(sizeV - 1);

  // Light sample direction vector polar coordinates. This is where the environment rotation happens!
  // DAR FIXME Use a light.matrix to rotate the resulting vector instead.
  const float phi   = (u - sysEnvironmentRotation) * 2.0f * M_PIf;
  const float theta = v * M_PIf; // theta == 0.0f is south pole, theta == M_PIf is north pole.

  const float sinTheta = sinf(theta);
  // The miss program places the 1->0 seam at the positive z-axis and looks from the inside.
  lightSample.direction = make_float3(-sinf(phi) * sinTheta,  // Starting on positive z-axis going around clockwise (to negative x-axis).
                                      -cosf(theta),           // From south pole to north pole.
                                       cosf(phi) * sinTheta); // Starting on positive z-axis.

  // Note that environment lights do not set the light sample position!
  lightSample.distance = RT_DEFAULT_MAX; // Environment light.

  const float3 emission = make_float3(optix::rtTex2D<float4>(light.idEnvironmentTexture, u, v));
  // Explicit light sample. The returned emission must be scaled by the inverse probability to select this light.
  lightSample.emission = emission * sysNumLights;
  // For simplicity we pretend that we perfectly importance-sampled the actual texture-filtered environment map
  // and not the Gaussian-smoothed one used to actually generate the CDFs and uniform sampling in the texel.
  lightSample.pdf = intensity(emission) / light.environmentIntegral;
}


RT_CALLABLE_PROGRAM void sample_light_parallelogram(float3 const& point, const float2 sample, LightSample& lightSample)
{
  lightSample.pdf = 0.0f; // Default return, invalid light sample (backface, edge on, or too near to the surface)

  const LightDefinition light = sysLightDefinitions[lightSample.index]; // The light index is picked by the caller!

  lightSample.position  = light.position + light.vecU * sample.x + light.vecV * sample.y; // The light sample position in world coordinates.
  lightSample.direction = lightSample.position - point; // Sample direction from surface point to light sample position.
  lightSample.distance  = optix::length(lightSample.direction);
  if (DENOMINATOR_EPSILON < lightSample.distance)
  {
    lightSample.direction /= lightSample.distance; // Normalized direction to light.

    const float cosTheta = optix::dot(-lightSample.direction, light.normal);
    if (DENOMINATOR_EPSILON < cosTheta) // Only emit light on the front side.
    {
      // Explicit light sample, must scale the emission by inverse probabilty to hit this light.
      lightSample.emission = light.emission * float(sysNumLights);
      lightSample.pdf      = (lightSample.distance * lightSample.distance) / (light.area * cosTheta); // Solid angle pdf. Assumes light.area != 0.0f.
    }
  }
}
