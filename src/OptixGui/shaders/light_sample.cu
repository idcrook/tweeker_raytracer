#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.cuh"
#include "per_ray_data.cuh"
#include "light_definition.cuh"
#include "shader_common.cuh"

#include "rt_assert.cuh"

rtBuffer<LightDefinition> sysLightDefinitions;
rtDeclareVariable(int,    sysNumLights, , ); // PERF Used many times and faster to read than sysLightDefinitions.size().


RT_FUNCTION void unitSquareToSphere(const float u, const float v, float3& p, float& pdf)
{
  p.z = 1.0f - 2.0f * u;
  float r = 1.0f - p.z * p.z;
  r = (0.0f < r) ? sqrtf(r) : 0.0f;

  const float phi = v * 2.0f * M_PIf;
  p.x = r * cosf(phi);
  p.y = r * sinf(phi);

  pdf = 0.25f * M_1_PIf;  // == 1.0f / (4.0f * M_PIf)
}

// Note that all light sampling routines return lightSample.direction and lightSample.distance in world space!

RT_CALLABLE_PROGRAM void sample_light_constant(float3 const& point, const float2 sample, LightSample& lightSample)
{
  unitSquareToSphere(sample.x, sample.y, lightSample.direction, lightSample.pdf);

  // Environment lights do not set the light sample position!
  lightSample.distance = RT_DEFAULT_MAX; // Environment light.

  // Explicit light sample. White scaled by inverse probabilty to hit this light.
  lightSample.emission = make_float3(sysNumLights);
}


RT_CALLABLE_PROGRAM void sample_light_parallelogram(float3 const& point, const float2 sample, LightSample& lightSample)
{
  lightSample.pdf = 0.0f; // Default return, invalid light sample (backface, edge on, or too near to the surface)

  const LightDefinition light = sysLightDefinitions[lightSample.index]; // The light index is picked by the caller!

  lightSample.position  = light.position + light.vecU * sample.x + light.vecV * sample.y; // The light sample position in world coordinates.
  lightSample.direction = lightSample.position - point; // Sample direction from surface point to light sample position.
  lightSample.distance  = optix::length(lightSample.direction);
  if (DENOMINATOR_EPSILON < lightSample.distance)
  {
    lightSample.direction /= lightSample.distance; // Normalized direction to light.

    const float cosTheta = optix::dot(-lightSample.direction, light.normal);
    if (DENOMINATOR_EPSILON < cosTheta) // Only emit light on the front side.
    {
      // Explicit light sample, must scale the emission by inverse probabilty to hit this light.
      lightSample.emission = light.emission * float(sysNumLights);
      lightSample.pdf      = (lightSample.distance * lightSample.distance) / (light.area * cosTheta); // Solid angle pdf. Assumes light.area != 0.0f.
    }
  }
}
