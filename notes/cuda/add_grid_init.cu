
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to initialize the elements of two arrays
__global__ void init(int n, float *x, float *y) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
}

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements

  // Allocate Unified Memory -- accessible from CPU or GPU
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  int blockSize = 256;
  //int blockSize = 512;
  //int blockSize = 320;
  //int blockSize = 32 * 4;
  int numBlocks = (N + blockSize - 1) / blockSize;

  init <<<numBlocks, blockSize>>>(N, x, y);
  add  <<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
