
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to add the elements of two arrays
// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements

// Allocate Unified Memory -- accessible from CPU or GPU
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  // float *x = new float[N];
  // float *y = new float[N];

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
  //add(N, x, y);
  // Run kernel on 1M elements on the GPU
  //add<<<1, 1>>>(N, x, y);
  add<<<1, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  // Free memory
  // delete [] x;
  // delete [] y;

  //cudaProfilerStop();

  return 0;
}
