#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.cuh"
#include "material_parameter.cuh"
#include "per_ray_data.cuh"
#include "shader_common.cuh"

rtDeclareVariable(optix::Ray, theRay,                  rtCurrentRay, );
rtDeclareVariable(float,      theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData,        thePrd,       rtPayload, );
rtDeclareVariable(PerRayData_shadow, thePrdShadow, rtPayload, );

// Attributes.
//rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
//rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

// Material parameter definition.
rtBuffer<MaterialParameter> sysMaterialParameters; // Context global buffer with an array of structures of MaterialParameter.
rtDeclareVariable(int,      parMaterialIndex, , ); // Per Material index into the above sysMaterialParameters array.


// One anyhit program for the radiance ray for all materials with cutout opacity!
RT_PROGRAM void anyhit_cutout() // For the radiance ray type.
{
  float opacity = 1.0f;
  const int id = sysMaterialParameters[parMaterialIndex].cutoutID; // Fetch the bindless texture ID for cutout opacity.
  if (id != RT_TEXTURE_ID_NULL)
  {
    opacity = intensity(make_float3(optix::rtTex2D<float4>(id, varTexCoord.x, varTexCoord.y))); // RGB intensity defines the opacity. White is opaque.
  }

  // Stochastic alpha test to get an alpha blend effect.
  if (opacity < 1.0f && opacity <= rng(thePrd.seed)) // No need to calculate an expensive random number if the test is going to fail anyway.
  {
    rtIgnoreIntersection();
  }
}


// The shadow ray program for all materials with no cutout opacity.
RT_PROGRAM void anyhit_shadow()
{
  thePrdShadow.visible = false;
  rtTerminateRay();
}

RT_PROGRAM void anyhit_shadow_cutout() // For the shadow ray type.
{
  float opacity = 1.0f;
  const int id = sysMaterialParameters[parMaterialIndex].cutoutID; // Fetch the bindless texture ID for cutout opacity.
  if (id != RT_TEXTURE_ID_NULL)
  {
    opacity = intensity(make_float3(optix::rtTex2D<float4>(id, varTexCoord.x, varTexCoord.y))); // RGB intensity defines the opacity. White is opaque.
  }

  // Stochastic alpha test to get an alpha blend effect.
  if (opacity < 1.0f && opacity <= rng(thePrdShadow.seed)) // No need to calculate an expensive random number if the test is going to fail anyway.
  {
    rtIgnoreIntersection();
  }
  else
  {
    thePrdShadow.visible = false;
    rtTerminateRay();
  }
}
