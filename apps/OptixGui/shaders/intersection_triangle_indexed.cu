#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "vertex_attributes.cuh"

rtBuffer<VertexAttributes> attributesBuffer;
rtBuffer<uint3>            indicesBuffer;

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

// Intersection routine for indexed interleaved triangle data.
RT_PROGRAM void intersection_triangle_indexed(int primitiveIndex)
{
  const uint3 indices = indicesBuffer[primitiveIndex];

  VertexAttributes const& a0 = attributesBuffer[indices.x];
  VertexAttributes const& a1 = attributesBuffer[indices.y];
  VertexAttributes const& a2 = attributesBuffer[indices.z];

  const float3 v0 = a0.vertex;
  const float3 v1 = a1.vertex;
  const float3 v2 = a2.vertex;

  float3 n;
  float  t;
  float  beta;
  float  gamma;

  if (intersect_triangle(theRay, v0, v1, v2, n, t, beta, gamma))
  {
    if (rtPotentialIntersection(t))
    {
      // Barycentric interpolation:
      const float alpha = 1.0f - beta - gamma;

      // Note: No normalization on the TBN attributes here for performance reasons.
      //       It's done after the transformation into world space anyway.
      varGeoNormal      = n;
      varTangent        = a0.tangent  * alpha + a1.tangent  * beta + a2.tangent  * gamma;
      varNormal         = a0.normal   * alpha + a1.normal   * beta + a2.normal   * gamma;
      varTexCoord       = a0.texcoord * alpha + a1.texcoord * beta + a2.texcoord * gamma;

      rtReportIntersection(0);
    }
  }
}
