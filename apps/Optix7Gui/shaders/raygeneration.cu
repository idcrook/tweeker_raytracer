#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.h"

#include <optix.h>

#include "system_parameter.h"
#include "per_ray_data.h"
#include "shader_common.h"
#include "random_number_generators.h"
#include "half_common.h"

extern "C" __constant__ SystemParameter sysParameter;

extern "C" __global__ void __raygen__pathtracer()
{
  PerRayData prd;

  const uint3 theLaunchDim   = optixGetLaunchDimensions(); // DAR This assumes that the launch dimentions are matching the size of the output buffer.
  const uint3 theLaunchIndex = optixGetLaunchIndex();

  // Initialize the random number generator seed from the linear pixel index and the iteration index.
  prd.seed = tea<4>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, sysParameter.iterationIndex);

  // DAR Decoupling the pixel coordinates from the screen size will allow for partial rendering algorithms.
  // In this case theLaunchIndex is the pixel coordinate and theLaunchDim is sysOutputBuffer.size().
  const float2 screen = make_float2(theLaunchDim);
  const float2 pixel  = make_float2(theLaunchIndex);
  const float2 sample = rng2(prd.seed);

  // Lens shaders
  optixDirectCall<void, const float2, const float2, const float2, float3&, float3&>(sysParameter.cameraType, screen, pixel, sample, prd.pos, prd.wi);

  // This renderer supports nested volumes. Four levels is plenty enough for most cases.
  // The absorption coefficient and IOR of the volume the ray is currently inside.
  float4 absorptionStack[MATERIAL_STACK_SIZE]; // .xyz == absorptionCoefficient (sigma_a), .w == index of refraction
  
  float3 radiance   = make_float3(0.0f); // Start with black.
  float3 throughput = make_float3(1.0f); // The throughput for the next radiance, starts with 1.0f.

#if USE_DENOISER_ALBEDO
  float3 albedo = make_float3(0.0f); // Start with black.
#endif
#if USE_DENOISER_NORMAL
  float3 normal = make_float3(0.0f); // Start with null vector.
  prd.normal    = make_float3(0.0f); // Start with null vector. Important if nothing is hit!
#endif
  
  int stackIdx = MATERIAL_STACK_EMPTY; // Start with empty nested materials stack.

  // Russian Roulette path termination after a specified number of bounces needs the current depth.
  int depth = 0; // Path segment index. Primary ray is 0. 

  prd.absorption_ior = make_float4(0.0f, 0.0f, 0.0f, 1.0f); // Assume primary ray starts in vacuum.
  prd.flags          = 0;

  while (depth < sysParameter.pathLengths.y)
  {
    prd.wo        = -prd.wi;            // Direction to observer.
    prd.ior       = make_float2(1.0f);  // Reset the volume IORs.
    prd.distance  = RT_DEFAULT_MAX;     // Shoot the next ray with maximum length.
    prd.flags    &= FLAG_CLEAR_MASK;    // Clear all non-persistent flags. In this demo only the last diffuse surface interaction stays.

    // Handle volume absorption of nested materials.
    if (MATERIAL_STACK_FIRST <= stackIdx) // Inside a volume?
    {
      prd.flags     |= FLAG_VOLUME;                            // Indicate that we're inside a volume. => At least absorption calculation needs to happen.
      prd.extinction = make_float3(absorptionStack[stackIdx]); // There is only volume absorption in this demo, no volume scattering.
      prd.ior.x      = absorptionStack[stackIdx].w;            // The IOR of the volume we're inside. Needed for eta calculations in transparent materials.
      if (MATERIAL_STACK_FIRST <= stackIdx - 1)
      {
        prd.ior.y = absorptionStack[stackIdx - 1].w; // The IOR of the surrounding volume. Needed when potentially leaving a volume to calculate eta in transparent materials.
      }
    }

    // Note that the primary rays (or volume scattering miss cases) wouldn't normally offset the ray t_min by sysSceneEpsilon. Keep it simple here.

    // Put radiance payload pointer into two unsigned integers.
    uint2 payload = splitPointer(&prd);

    optixTrace(sysParameter.topObject,
               prd.pos, prd.wi, // origin, direction
               sysParameter.sceneEpsilon, prd.distance, 0.0f, // tmin, tmax, time
               OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_NONE, 
               RAYTYPE_RADIANCE, NUM_RAYTYPES, RAYTYPE_RADIANCE,
               payload.x, payload.y);

    // This renderer supports nested volumes.
    if (prd.flags & FLAG_VOLUME)
    {
      // We're inside a volume. Calculate the extinction along the current path segment in any case.
      // The transmittance along the current path segment inside a volume needs to attenuate the ray throughput with the extinction
      // before it modulates the radiance of the hitpoint.
      throughput *= expf(-prd.distance * prd.extinction);
    }

    radiance += throughput * prd.radiance;

#if USE_DENOISER_ALBEDO
    // In physical terms, the albedo is a single color value approximating the ratio of radiant exitance to the irradiance under uniform lighting.
    // The albedo value can be approximated for simple materials by using the diffuse color of the first hit,
    // or for layered materials by using a weighted sum of the individual BRDFs albedo values.
    // For some objects such as perfect mirrors, the quality of the result might be improved by using the albedo value of a subsequent hit instead.

    // When no albedo has been written before and the hit was diffuse or a light, write the albedo.
    // DAR This makes glass materials and motion blur on specular surfaces in the demo a little noisier,
    // but should definitely be used with high frequency textures behind transparent or around reflective materials.
    if (!(prd.flags & FLAG_ALBEDO) && (prd.flags & (FLAG_DIFFUSE | FLAG_LIGHT)))
    {
      // The albedo buffer should contain the surface appearance under uniform lighting in linear color space in the range [0.0f, 1.0f].
      // Clamp the final albedo result to that range here, because it captured the radiance when hitting lights either directly or via specular events.
      albedo = clamp(throughput * prd.albedo, 0.0f, 1.0f);

      prd.flags |= FLAG_ALBEDO; // This flag is persistent along the path and prevents that the albedo is written more than once.
    }
#endif 

#if USE_DENOISER_NORMAL
    // The normal buffer is expected to contain the surface normals of the primary hit in camera space.
    // The camera space is assumed to be right handed such that the camera is looking down
    // the negative z-axis, and the up direction is along the y-axis. The x-axis points to the right.
    if (depth == 0 && (prd.flags & FLAG_HIT)) // Miss event keeps the null-vector.
    {
      // Note the input sysCameraU|V|W vectors are unnormalized and build a left-handed coordinate system.
      // They are also not necessarily perpendicular to each other, because the generic pinhole camera system
      // would allow sheared projections, but that's not used on these OptiX introduction examples. 
      
      // Project the world space normal into camera space.
      // Using the normalized camera basis vectors here as camera space to get consistent results
      // independently of the UVW vector lengths.
      // The end result looks like a normal map without scale and bias.
      // Normals pointing at the camera position will be blue.
      normal = make_float3( dot(prd.normal, normalize(sysParameter.cameraU)), 
                            dot(prd.normal, normalize(sysParameter.cameraV)), 
                           -dot(prd.normal, normalize(sysParameter.cameraW))); // Negative W to make it right-handed.
    }
#endif

    // Path termination by miss shader or sample() routines.
    // If terminate is true, f_over_pdf and pdf might be undefined.
    if ((prd.flags & FLAG_TERMINATE) || prd.pdf <= 0.0f || isNull(prd.f_over_pdf))
    {
      break;
    }

    // PERF f_over_pdf already contains the proper throughput adjustment for diffuse materials: f * (fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf);
    throughput *= prd.f_over_pdf;

    // Unbiased Russian Roulette path termination.
    if (sysParameter.pathLengths.x <= depth) // Start termination after a minimum number of bounces.
    {
      const float probability = fmaxf(throughput); // DAR Other options: // intensity(throughput); // fminf(0.5f, intensity(throughput));
      if (probability < rng(prd.seed)) // Paths with lower probability to continue are terminated earlier.
      {
        break;
      }
      throughput /= probability; // Path isn't terminated. Adjust the throughput so that the average is right again.
    }

    // Adjust the material volume stack if the geometry is not thin-walled but a border between two volumes 
    // and the outgoing ray direction was a transmission.
    if ((prd.flags & (FLAG_THINWALLED | FLAG_TRANSMISSION)) == FLAG_TRANSMISSION) 
    {
      // Transmission.
      if (prd.flags & FLAG_FRONTFACE) // Entered a new volume?
      {
        // Push the entered material's volume properties onto the volume stack.
        //rtAssert((stackIdx < MATERIAL_STACK_LAST), 1); // Overflow?
        stackIdx = min(stackIdx + 1, MATERIAL_STACK_LAST);
        absorptionStack[stackIdx] = prd.absorption_ior;
      }
      else // Exited the current volume?
      {
        // Pop the top of stack material volume.
        // This assert fires and is intended because I tuned the frontface checks so that there are more exits than enters at silhouettes.
        //rtAssert((MATERIAL_STACK_EMPTY < stackIdx), 0); // Underflow?
        stackIdx = max(stackIdx - 1, MATERIAL_STACK_EMPTY);
      }
    }

    ++depth; // Next path segment.
  }

#if USE_DEBUG_EXCEPTIONS
  // DAR DEBUG Highlight numerical errors.
  if (isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z))
  {
    radiance = make_float3(1000000.0f, 0.0f, 0.0f); // super red
  }
  else if (isinf(radiance.x) || isinf(radiance.y) || isinf(radiance.z))
  {
    radiance = make_float3(0.0f, 1000000.0f, 0.0f); // super green
  }
  else if (radiance.x < 0.0f || radiance.y < 0.0f || radiance.z < 0.0f)
  {
    radiance = make_float3(0.0f, 0.0f, 1000000.0f); // super blue
  }
#else
  // NaN values will never go away. Filter them out before they can arrive in the output buffer.
  // This only has an effect if the debug coloring above is off!
  if (!(isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z)))
#endif
  {
    const unsigned int index = theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x;

#if USE_FP32_OUTPUT

    float4* bufferRGBA   = reinterpret_cast<float4*>(sysParameter.outputBuffer);
#if USE_DENOISER_ALBEDO
    float4* bufferAlbedo = reinterpret_cast<float4*>(sysParameter.albedoBuffer);
#endif
#if USE_DENOISER_NORMAL
    float4* bufferNormal = reinterpret_cast<float4*>(sysParameter.normalBuffer);
#endif
    if (0 < sysParameter.iterationIndex)
    {
      const float t = 1.0f / float(sysParameter.iterationIndex + 1);

      radiance = lerp(make_float3(bufferRGBA[index]),   radiance, t);
#if USE_DENOISER_ALBEDO
      albedo   = lerp(make_float3(bufferAlbedo[index]), albedo,   t);
#endif
#if USE_DENOISER_NORMAL
      normal   = lerp(make_float3(bufferNormal[index]), normal,   t);
      if (isNotNull(normal))
      {
        normal = normalize(normal);
      }
#endif
    }
    // sysIterationIndex 0 will fill the buffer.
    // If this isn't done separately, the result of the lerp() above is undefined, e.g. dst could be NaN.
    bufferRGBA[index]   = make_float4(radiance, 1.0f);
#if USE_DENOISER_ALBEDO
    bufferAlbedo[index] = make_float4(albedo,   1.0f);
#endif
#if USE_DENOISER_NORMAL
    bufferNormal[index] = make_float4(normal,   0.0f);
#endif

#else

    Half4* bufferRGBA   = reinterpret_cast<Half4*>(sysParameter.outputBuffer);
#if USE_DENOISER_ALBEDO
    Half4* bufferAlbedo = reinterpret_cast<Half4*>(sysParameter.albedoBuffer);
#endif
#if USE_DENOISER_NORMAL
    Half4* bufferNormal = reinterpret_cast<Half4*>(sysParameter.normalBuffer);
#endif
    
    if (0 < sysParameter.iterationIndex)
    {
      const float t = 1.0f / float(sysParameter.iterationIndex + 1);
      
      Half4 dst = bufferRGBA[index]; // RGBA16F

      radiance.x = lerp(__half2float(dst.x), radiance.x, t);
      radiance.y = lerp(__half2float(dst.y), radiance.y, t);
      radiance.z = lerp(__half2float(dst.z), radiance.z, t);

#if USE_DENOISER_ALBEDO
      dst = bufferAlbedo[index]; // RGBA16F

      albedo.x = lerp(__half2float(dst.x), albedo.x, t);
      albedo.y = lerp(__half2float(dst.y), albedo.y, t);
      albedo.z = lerp(__half2float(dst.z), albedo.z, t);
#endif
#if USE_DENOISER_NORMAL
      dst = bufferNormal[index]; // RGBA16F

      normal.x = lerp(__half2float(dst.x), normal.x, t);
      normal.y = lerp(__half2float(dst.y), normal.y, t);
      normal.z = lerp(__half2float(dst.z), normal.z, t);
      if (isNotNull(normal))
      {
        normal = normalize(normal);
      }
#endif
    }
    // sysIterationIndex 0 will fill the buffer.
    // If this isn't done separately, the result of the lerp() above is undefined, e.g. dst could be NaN.
    bufferRGBA[index]   = make_Half4(radiance, 1.0f);
#if USE_DENOISER_ALBEDO
    bufferAlbedo[index] = make_Half4(albedo,   1.0f);
#endif
#if USE_DENOISER_NORMAL
    bufferNormal[index] = make_Half4(normal,   0.0f);
#endif

#endif
  }
}
